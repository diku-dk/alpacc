#include "hip/hip_runtime.h"
__device__ __host__ __forceinline__
state_t get_index(state_t state) {
  return (state & ENDO_MASK) >> ENDO_OFFSET;
}

__device__ __host__ __forceinline__
token_t get_token(state_t state) {
  return (state & TOKEN_MASK) >> TOKEN_OFFSET;
}

__device__ __host__ __forceinline__
bool is_produce(state_t state) {
  return (state & PRODUCE_MASK) >> PRODUCE_OFFSET;
}

state_t get_index_cpu(state_t state) {
  return (state & ENDO_MASK) >> ENDO_OFFSET;
}

token_t get_token_cpu(state_t state) {
  return (state & TOKEN_MASK) >> TOKEN_OFFSET;
}

bool is_produce_cpu(state_t state) {
  return (state & PRODUCE_MASK) >> PRODUCE_OFFSET;
}

template<typename T>
struct TakeRight {
  const T identity = std::numeric_limits<T>::max();

  __device__ __forceinline__ T operator()(T a, T b) const {
    if (b == identity) {
      return a;
    }

    return b;
  }
};

template<typename I, typename J>
struct LexerCtx {

private:
  J offset = 0;
  state_t* d_to_state;
  state_t* d_compose;
  volatile unsigned int* d_dyn_block_index;
  volatile state_t* d_new_last_state;
  volatile state_t* d_old_last_state;
  I* d_new_size;
  volatile J* d_new_last_start;
  volatile J* d_old_last_start;

  void swapLastStart() {
    J h_last_start;
    gpuAssert(hipMemcpy(&h_last_start, (const void*) d_new_last_start, sizeof(J), hipMemcpyDeviceToHost));
    gpuAssert(hipMemcpy((void *) d_new_last_start, (const void*) d_old_last_start, sizeof(J), hipMemcpyDeviceToDevice));
    gpuAssert(hipMemcpy((void *) d_old_last_start, &h_last_start, sizeof(J), hipMemcpyHostToDevice));
  }

  void swapLastState() {
  state_t h_last_state;
  gpuAssert(hipMemcpy(&h_last_state, (const void*) d_new_last_state, sizeof(state_t), hipMemcpyDeviceToHost));
  gpuAssert(hipMemcpy((void *) d_new_last_state, (const void*) d_old_last_state, sizeof(state_t), hipMemcpyDeviceToDevice));
  gpuAssert(hipMemcpy((void *) d_old_last_state, &h_last_state, sizeof(state_t), hipMemcpyHostToDevice));
}

  void resetDynamicIndex() const {
    hipMemset((void*)d_dyn_block_index, 0, sizeof(unsigned int));
  }

  void updateOffset() {
    offset += CHUNK_SIZE;
  }

  void resetNewSize() const {
    hipMemset(d_new_size, 0, sizeof(I));
  }

public:
  const I CHUNK_SIZE;
  States<I, state_t> d_state_states;
  States<I, I> d_index_states;
  States<I, I> d_take_right_states;
  TakeRight<I> take_right = TakeRight<I>();

  LexerCtx(const I chunk_size,
           const I block_size,
           const I items_per_thread) : CHUNK_SIZE(chunk_size) {
    I num_blocks = numBlocks(chunk_size, block_size, items_per_thread);
    gpuAssert(hipMalloc(&d_to_state, sizeof(h_to_state)));
    hipMemcpy(d_to_state, h_to_state, sizeof(h_to_state),
                 hipMemcpyHostToDevice);
    gpuAssert(hipMalloc(&d_compose, sizeof(h_compose)));
    hipMemcpy(d_compose, h_compose, sizeof(h_compose),
                 hipMemcpyHostToDevice);

    d_index_states = States<I, I>(num_blocks);
    d_take_right_states = States<I, I>(num_blocks);
    d_state_states = States<I, state_t>(num_blocks);

    gpuAssert(hipMalloc((void**)&d_dyn_block_index, sizeof(unsigned int)));
    gpuAssert(hipMalloc((void**)&d_new_size, sizeof(I)));
    gpuAssert(hipMalloc((void**)&d_new_last_state, sizeof(state_t)));
    gpuAssert(hipMalloc((void**)&d_old_last_state, sizeof(state_t)));
    gpuAssert(hipMalloc((void**)&d_new_last_start, sizeof(J)));
    gpuAssert(hipMalloc((void**)&d_old_last_start, sizeof(J)));

    hipMemset((void*)d_dyn_block_index, 0, sizeof(unsigned int));
    hipMemset((void*)d_new_size, I(), sizeof(I));
    hipMemset((void*)d_new_last_state, IDENTITY, sizeof(state_t));
    hipMemset((void*)d_old_last_state, IDENTITY, sizeof(state_t));
    hipMemset((void*)d_new_last_start, J(), sizeof(J));
    hipMemset((void*)d_old_last_start, J(), sizeof(J));
  }

  void cleanUp() {
    if (d_to_state) hipFree(d_to_state);
    if (d_new_last_start) hipFree((void*)d_new_last_start);
    if (d_old_last_start) hipFree((void*)d_old_last_start);
    if (d_compose) hipFree(d_compose);
    if (d_dyn_block_index) hipFree((void*)d_dyn_block_index);
    if (d_new_size) hipFree((void*)d_new_size);
    if (d_new_last_state) hipFree((void*)d_new_last_state);
    if (d_old_last_state) hipFree((void*)d_old_last_state);
    d_index_states.cleanUp();
    d_state_states.cleanUp();
    d_take_right_states.cleanUp();
  }

  __device__ __host__ __forceinline__
  state_t operator()(const state_t &a, const state_t &b) const {
    return d_compose[get_index(b) * NUM_STATES + get_index(a)];
  }

  __device__ __host__ __forceinline__
  state_t operator()(const volatile state_t &a, const volatile state_t &b) const {
    return d_compose[get_index(b) * NUM_STATES + get_index(a)];
  }

  __device__ __host__ __forceinline__
  state_t toState(const char &a) const {
    return d_to_state[a];
  }

  __device__ __host__ __forceinline__
  J addOffset(I i) const {
    return i + offset;
  }

  __device__ __forceinline__
  unsigned int getDynamicIndex() const {
    return dynamicIndex(d_dyn_block_index);
  }

  __device__ __host__ __forceinline__
  void setLastState(state_t state) const {
    *d_new_last_state = state;
  }

  __device__ __host__ __forceinline__
  state_t getLastState() const {
    return *d_old_last_state;
  }

  __device__ __host__ __forceinline__
  void setNewSize(I size) const {
    *d_new_size = size;
  }

  __device__ __host__ __forceinline__
  void setLastStart(J i) const {
    *d_new_last_start = i;
  }

  __device__ __host__ __forceinline__
  J getLastStart() const {
    return *d_old_last_start;
  }

  bool isAccept() const {
    state_t h_last_state;
    gpuAssert(hipMemcpy(&h_last_state, (const void*) d_new_last_state, sizeof(state_t), hipMemcpyDeviceToHost));
    return h_accept[get_index_cpu(h_last_state)];
  }

  I tokensSize() const {
    I h_new_size = I();
    gpuAssert(hipMemcpy(&h_new_size, (const void*) d_new_size, sizeof(I), hipMemcpyDeviceToHost));
    return h_new_size;
  }

  void update() {
    resetDynamicIndex();
    swapLastStart();
    swapLastState();
    updateOffset();
  }
};

template<typename I, typename J, I BLOCK_SIZE, I ITEMS_PER_THREAD>
__global__ void
lexer(LexerCtx<I, J> ctx, unsigned char* d_string, token_t* d_tokens, J* d_starts, J* d_ends, const I size, const bool is_last_chunk) {
  volatile __shared__ state_t states[ITEMS_PER_THREAD * BLOCK_SIZE];
  volatile __shared__ I indices[ITEMS_PER_THREAD * BLOCK_SIZE];
  volatile __shared__ I indices_aux[BLOCK_SIZE];
  __shared__ state_t next_block_first_state;
  volatile state_t* states_aux = (volatile state_t*) indices;
  const I REG_MEM = 1 + ITEMS_PER_THREAD / sizeof(unsigned long long);
  unsigned long long copy_reg[REG_MEM];
  unsigned char *chars_reg = (unsigned char*) copy_reg;
  unsigned int is_produce_state = 0;

  unsigned int dyn_index = ctx.getDynamicIndex();
  I glb_offs = dyn_index * BLOCK_SIZE * ITEMS_PER_THREAD;

  if (threadIdx.x == I()) {
    next_block_first_state = IDENTITY;
  }

#pragma unroll
  for (I i = 0; i < REG_MEM; i++) {
    I uint64_lid = i * blockDim.x + threadIdx.x;
    I lid = sizeof(unsigned long long) * uint64_lid;
    I gid = glb_offs + lid;
    if (gid + sizeof(unsigned long long) < size) {
      copy_reg[i] = *((unsigned long long*) (gid + (unsigned char*) d_string));
    } else {
      for (I j = 0; j < sizeof(unsigned long long); j++) {
        I loc_gid = gid + j;
        if (loc_gid < size) {
          chars_reg[sizeof(unsigned long long) * i + j] = d_string[loc_gid];
        }
      }
    }
  }
    
#pragma unroll
  for (I i = 0; i < REG_MEM; i++) {
    I lid = i * blockDim.x + threadIdx.x;
    I _gid = glb_offs + sizeof(unsigned long long) * lid;
    for (I j = 0; j < sizeof(unsigned long long); j++) {
      I gid = _gid + j;
      I lid_off = sizeof(unsigned long long) * lid + j;
      I reg_off = sizeof(unsigned long long) * i + j;
      bool is_in_block = lid_off < ITEMS_PER_THREAD * BLOCK_SIZE; 
      if (gid < size && is_in_block) {
          if (gid == 0) {
            states[lid_off] = ctx(ctx.getLastState(), reinterpret_cast<state_t>(ctx.toState(chars_reg[reg_off])));
          } else {
            states[lid_off] = ctx.toState(chars_reg[reg_off]);
          }
      } else if (is_in_block) {
          states[lid_off] = IDENTITY;
      } else if (lid_off == ITEMS_PER_THREAD * BLOCK_SIZE) {
          next_block_first_state = ctx.toState(chars_reg[reg_off]);
      }
    }
  }

  __syncthreads();

  scan<state_t, I, LexerCtx<I, J>, ITEMS_PER_THREAD>(states, states_aux, ctx.d_state_states, ctx, IDENTITY, dyn_index);

#pragma unroll
  for (I i = 0; i < ITEMS_PER_THREAD; i++) {
    I lid = i * blockDim.x + threadIdx.x;
    I gid = glb_offs + lid;
    bool is_next_produce = false;
    if (gid < size) {
      state_t state = states[lid];
#ifdef IGNORE_TOKEN
      bool is_not_ignore = get_token(state) != IGNORE_TOKEN;
#else
      bool is_not_ignore = true;
#endif
      if (lid == ITEMS_PER_THREAD * BLOCK_SIZE - 1) {
        is_next_produce = is_produce(ctx(state, next_block_first_state));
      } else {
        is_next_produce = is_produce(states[lid + 1]);
      }

      if (is_last_chunk) {
        is_next_produce |= gid == size - 1;
        is_next_produce &= is_not_ignore;
      } else {
        is_next_produce &= is_not_ignore && gid != size - 1;
      }

      indices[lid] = is_produce(state) ? gid : ctx.take_right.identity;
    } else {
      indices[lid] = ctx.take_right.identity;
    }
    is_produce_state |= is_next_produce << i;
  }

  __syncthreads();

  scan<I, I, TakeRight<I>, ITEMS_PER_THREAD>(indices, indices_aux, ctx.d_take_right_states, ctx.take_right, ctx.take_right.identity, dyn_index);

  I starts[ITEMS_PER_THREAD];
  volatile __shared__ I last_start;

#pragma unroll
  for (I i = 0; i < ITEMS_PER_THREAD; i++) {
    I lid = i * blockDim.x + threadIdx.x;
    I gid = glb_offs + lid;

    if (gid < size) {
      starts[i] = indices[lid];
      indices[lid] = (is_produce_state >> i) & 1;

      if (gid == size - 1) {
        last_start = starts[i];
      }
      
    } else {
      indices[lid] = 0;
    }
  }

  __syncthreads();

  I prefix = scan<I, I, Add<I>, ITEMS_PER_THREAD>(indices, indices_aux, ctx.d_index_states, Add<I>(), I(), dyn_index, false);

  #pragma unroll
  for (I i = 0; i < ITEMS_PER_THREAD; i++) {
    I lid = blockDim.x * i + threadIdx.x;
    I gid = glb_offs + lid;
    if (gid < size && ((is_produce_state >> i) & 1)) {
      I offset = Add<I>()(prefix, indices[lid]) - 1;
      if (offset == I() && starts[i] == ctx.take_right.identity) {
        d_starts[offset] = ctx.getLastStart();
      } else {
        d_starts[offset] = ctx.addOffset(starts[i]);
      }
      d_ends[offset] = ctx.addOffset(gid + 1);
      d_tokens[offset] = get_token(states[lid]);
    }
  }

  if (dyn_index == gridDim.x - 1 && threadIdx.x == blockDim.x - 1) {
    I new_size = Add<I>()(prefix, indices[ITEMS_PER_THREAD * BLOCK_SIZE - 1]);
    ctx.setNewSize(new_size);
    ctx.setLastState(states[ITEMS_PER_THREAD * BLOCK_SIZE - 1]);
    
    if (last_start != ctx.take_right.identity) {
      ctx.setLastStart(ctx.addOffset(last_start));
    } else {
      ctx.setLastStart(ctx.getLastStart());
    }
  }
}


struct WriteBinary {
  void operator()(size_t i, size_t j, token_t s) const {
    unsigned char* buffer[2 * sizeof(size_t) + sizeof(token_t)];
    memcpy(buffer, &i, sizeof(size_t));
    memcpy(buffer + sizeof(size_t), &j, sizeof(size_t));
    memcpy(buffer + 2 * sizeof(size_t), &s, sizeof(token_t));
    fwrite(buffer, 2 * sizeof(size_t) + sizeof(token_t), 1, stdout);
  }
};

struct WriteAscii {
  void operator()(size_t i, size_t j, token_t s) const {
    printf("%lu %lu %lu\n", i, j, (size_t) s);
  }
};


struct NoWrite {
  void operator()(size_t i, size_t j, token_t s) const {
    // No operation
  }
};

bool read_chunk(FILE* file, unsigned char* buffer, size_t chunk_size, size_t* bytes_read) {
    *bytes_read = fread(buffer, sizeof(unsigned char), chunk_size, file);
    
    bool is_not_done = (*bytes_read == chunk_size) && !feof(file);
    
    if (*bytes_read == chunk_size) {
      int next_char = fgetc(file);
      if (next_char != EOF) {
        ungetc(next_char, file);
        is_not_done = true;
      } else {
        is_not_done = false;
      }
    }
    
    return is_not_done;
}

template<typename PRINT>
int lexer_stream(PRINT print, bool timeit = false) {
  const unsigned int chunk_size = 64; // 100 * (1 << 20); // 100MiB
  
  unsigned char* h_string = (unsigned char*) malloc(chunk_size * sizeof(unsigned char));
  token_t* h_tokens = (token_t*) malloc(chunk_size * sizeof(token_t));
  size_t* h_starts = (size_t*) malloc(chunk_size * sizeof(size_t));
  size_t* h_ends = (size_t*) malloc(chunk_size * sizeof(size_t));
  assert(h_string != NULL);
  assert(h_tokens != NULL);
  assert(h_starts != NULL);
  assert(h_ends != NULL);

  unsigned char* d_string;
  token_t* d_tokens;
  size_t* d_starts;
  size_t* d_ends;
  gpuAssert(hipMalloc((void**)&d_string, chunk_size * sizeof(unsigned char)));
  gpuAssert(hipMalloc((void**)&d_tokens, chunk_size * sizeof(token_t)));
  gpuAssert(hipMalloc((void**)&d_starts, chunk_size * sizeof(size_t)));
  gpuAssert(hipMalloc((void**)&d_ends, chunk_size * sizeof(size_t)));

  const unsigned int BLOCK_SIZE = 32;
  assert(WARP <= BLOCK_SIZE);
  const unsigned int ITEMS_PER_THREAD = 1;
  LexerCtx ctx = LexerCtx<unsigned int, size_t>(chunk_size, BLOCK_SIZE, ITEMS_PER_THREAD);

  size_t new_size = 0;
  size_t final_size = 0;
  float time = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  bool is_not_done = true;
  while (is_not_done) {
    size_t bytes;
    is_not_done = read_chunk(stdin, h_string, chunk_size, &bytes);
    final_size += bytes;

    gpuAssert(hipMemcpy(d_string, h_string, bytes, hipMemcpyHostToDevice));

    const unsigned int num_blocks = numBlocks(bytes, BLOCK_SIZE, ITEMS_PER_THREAD);
    hipEventRecord(start, 0);
    lexer<unsigned int, size_t, BLOCK_SIZE, ITEMS_PER_THREAD><<<num_blocks, BLOCK_SIZE>>>(ctx, d_string, d_tokens, d_starts, d_ends, bytes, !is_not_done);
    gpuAssert(hipDeviceSynchronize());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float temp = 0;
    hipEventElapsedTime(&temp, start, stop);
    gpuAssert(hipPeekAtLastError());
    time += temp;

    new_size = ctx.tokensSize();

    hipMemcpy(h_tokens, d_tokens, new_size * sizeof(token_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_starts, d_starts, new_size * sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_ends, d_ends, new_size * sizeof(size_t), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < new_size; i++) {
      print(h_starts[i], h_ends[i], h_tokens[i]);
    }

    ctx.update();
  }

  if (timeit) {
    printf("Time: %.2fms\n", time);
  }

  fflush(stdout);

  int success = ctx.isAccept() ? 0 : -1;
  
  ctx.cleanUp();

  free(h_string);
  free(h_tokens);
  free(h_starts);
  hipFree(d_string);
  hipFree(d_tokens);
  hipFree(d_starts);
  return success;
}

template<unsigned int BLOCK_SIZE, unsigned int ITEMS_PER_THREAD>
bool lexer_full(
  LexerCtx<unsigned int, size_t> ctx, 
  unsigned char* d_string,
  token_t* d_tokens,
  size_t* d_starts,
  size_t* d_ends,
  size_t chunk_size, 
  size_t size,
  size_t* new_size) {
  assert(chunk_size <= size);
  assert(size != 0);
  assert(d_string != NULL);
  assert(d_tokens == NULL);
  assert(d_starts == NULL);
  assert(d_ends == NULL);
  assert(WARP <= BLOCK_SIZE);
  assert(ITEMS_PER_THREAD > 1);

  hipMalloc((void**)&d_tokens, chunk_size * sizeof(token_t));
  hipMalloc((void**)&d_starts, chunk_size * sizeof(size_t));
  hipMalloc((void**)&d_ends, chunk_size * sizeof(size_t));

  size_t prev_index = 0;
  size_t temp_new_size = 0;
  size_t alloc_size = chunk_size;
  float time = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for (size_t offset = 0; offset < size; offset+=chunk_size) {
    unsigned int bytes = min(chunk_size, size - offset);

    const unsigned int NUM_BLOCKS = numBlocks(bytes, BLOCK_SIZE, ITEMS_PER_THREAD);
    hipEventRecord(start, 0);
    lexer<unsigned int, size_t, BLOCK_SIZE, ITEMS_PER_THREAD><<<NUM_BLOCKS, BLOCK_SIZE>>>(ctx, d_string, d_tokens, d_starts, d_ends, bytes, offset < size - chunk_size);
    gpuAssert(hipDeviceSynchronize());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float temp = 0;
    hipEventElapsedTime(&temp, start, stop);
    gpuAssert(hipPeekAtLastError());
    time += temp;

    temp_new_size += ctx.tokensSize();

    if (alloc_size < temp_new_size + chunk_size) {
      while (alloc_size < temp_new_size + chunk_size) {
        alloc_size *= 2;
      }
      hipMalloc((void**)&d_tokens, alloc_size * sizeof(token_t));
      hipMalloc((void**)&d_starts, alloc_size * sizeof(size_t));
      hipMalloc((void**)&d_ends, alloc_size * sizeof(size_t));
    }

    ctx.update();
  }

  *new_size = temp_new_size;
  
  hipMalloc((void**)&d_tokens, temp_new_size * sizeof(token_t));
  hipMalloc((void**)&d_starts, temp_new_size * sizeof(size_t));
  hipMalloc((void**)&d_ends, temp_new_size * sizeof(size_t));

  return ctx.isAccept();
}
